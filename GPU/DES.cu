#include "hip/hip_runtime.h"
#include "DES.h"

#ifdef __CUDA_ARCH__
#define CONSTANT __constant__
#define DEVICE __device__
#else
#define CONSTANT
#define DEVICE
#endif

#include "hip/hip_runtime.h"
#include ""

namespace cuda::des
{
	CONSTANT unsigned char key_perm_l[56] = {
		3, 2, 1, 0, 3, 2, 1, 0,
		3, 2, 1, 0, 7, 6, 5, 4,
		3, 2, 1, 0, 7, 6, 5, 4,
		7, 6, 5, 4, 7, 6, 5, 4,
		7, 6, 5, 4, 3, 2, 1, 0,
		7, 6, 5, 4, 3, 2, 1, 0,
		7, 6, 5, 4, 3, 2, 1, 0,
	};

	CONSTANT unsigned char key_perm_r[56] = {
		8, 8, 8, 8, 16, 16, 16, 16,
		4, 4, 4, 4, 8, 8, 8, 8,
		2, 2, 2, 2, 4, 4, 4, 4,
		16, 16, 16, 16, 2, 2, 2, 2,
		32, 32, 32, 32, 32, 32, 32, 32,
		64, 64, 64, 64, 64, 64, 64, 64,
		128, 128, 128, 128, 128, 128, 128, 128,
	};

	CONSTANT unsigned char left_round_perm_l[24] = {
		3, 0, 1, 0, 3, 2, 1, 0,
		1, 0, 2, 1, 2, 2, 1, 0,
		1, 2, 1, 2, 0, 0, 0, 3,
	};

	CONSTANT unsigned char left_round_perm_r[24] = {
		64, 1, 1, 2, 32, 16, 8, 64,
		2, 4, 8, 64, 2, 32, 16, 16,
		4, 128, 32, 1, 128, 8, 32, 16,
	};

	CONSTANT unsigned char right_round_perm_l[24] = {
		4, 6, 5, 5, 6, 4, 3, 3,
		6, 5, 4, 5, 5, 6, 4, 6,
		5, 6, 3, 4, 5, 6, 3, 4,
	};

	CONSTANT unsigned char right_round_perm_r[24] = {
		64, 8, 4, 64, 64, 16, 8, 1,
		32, 8, 128, 1, 16, 128, 2, 1,
		128, 16, 2, 8, 2, 2, 4, 1,
	};

	CONSTANT unsigned char initial_perm_l[64] = {
		0, 1, 2, 3, 4, 5, 6, 7,
		0, 1, 2, 3, 4, 5, 6, 7,
		0, 1, 2, 3, 4, 5, 6, 7,
		0, 1, 2, 3, 4, 5, 6, 7,
		0, 1, 2, 3, 4, 5, 6, 7,
		0, 1, 2, 3, 4, 5, 6, 7,
		0, 1, 2, 3, 4, 5, 6, 7,
		0, 1, 2, 3, 4, 5, 6, 7,
	};

	CONSTANT unsigned char initial_perm_r[64] = {
		64, 64, 64, 64, 64, 64, 64, 64,
		16, 16, 16, 16, 16, 16, 16, 16,
		4, 4, 4, 4, 4, 4, 4, 4,
		1, 1, 1, 1, 1, 1, 1, 1,
		128, 128, 128, 128, 128, 128, 128, 128,
		32, 32, 32, 32, 32, 32, 32, 32,
		8, 8, 8, 8, 8, 8, 8, 8,
		2, 2, 2, 2, 2, 2, 2, 2,
	};

	CONSTANT unsigned char final_perm_l[64] = {
		3, 7, 2, 6, 1, 5, 0, 4,
		3, 7, 2, 6, 1, 5, 0, 4,
		3, 7, 2, 6, 1, 5, 0, 4,
		3, 7, 2, 6, 1, 5, 0, 4,
		3, 7, 2, 6, 1, 5, 0, 4,
		3, 7, 2, 6, 1, 5, 0, 4,
		3, 7, 2, 6, 1, 5, 0, 4,
		3, 7, 2, 6, 1, 5, 0, 4,
	};

	CONSTANT unsigned char final_perm_r[64] = {
		1, 1, 1, 1, 1, 1, 1, 1,
		2, 2, 2, 2, 2, 2, 2, 2,
		4, 4, 4, 4, 4, 4, 4, 4,
		8, 8, 8, 8, 8, 8, 8, 8,
		16, 16, 16, 16, 16, 16, 16, 16,
		32, 32, 32, 32, 32, 32, 32, 32,
		64, 64, 64, 64, 64, 64, 64, 64,
		128, 128, 128, 128, 128, 128, 128, 128,
	};

	CONSTANT unsigned char expansion_l[48] = {
		0, 0, 0, 0, 0, 0, 0, 3,
		1, 1, 1, 0, 1, 0, 0, 0,
		2, 1, 1, 1, 1, 1, 1, 1,
		2, 2, 2, 2, 2, 2, 2, 1,
		3, 3, 3, 2, 3, 2, 2, 2,
		0, 3, 3, 3, 3, 3, 3, 3,
	};

	CONSTANT unsigned char expansion_r[48] = {
		8, 16, 8, 16, 32, 64, 128, 1,
		32, 64, 128, 1, 128, 1, 2, 4,
		128, 1, 2, 4, 8, 16, 8, 16,
		8, 16, 8, 16, 32, 64, 128, 1,
		32, 64, 128, 1, 128, 1, 2, 4,
		128, 1, 2, 4, 8, 16, 8, 16,
	};

	CONSTANT unsigned char pbox_l[32] = {
		2, 3, 1, 3, 2, 2, 0, 1,
		1, 3, 2, 0, 3, 2, 1, 0,
		1, 0, 3, 3, 1, 2, 0, 0,
		3, 0, 1, 2, 0, 3, 1, 2,
	};

	CONSTANT unsigned char pbox_r[32] = {
		128, 16, 16, 8, 8, 16, 2, 1,
		64, 2, 64, 8, 64, 2, 2, 128,
		128, 32, 32, 1, 4, 1, 1, 64,
		128, 16, 32, 4, 4, 4, 8, 32,
	};

	CONSTANT unsigned char sbox[2][4][4][16] = {
		14,	4,	13,	1,	2,	15,	11,	8,	3,	10,	6,	12,	5,	9,	0,	7,
		0,	15,	7,	4,	14,	2,	13,	1,	10,	6,	12,	11,	9,	5,	3,	8,
		4,	1,	14,	8,	13,	6,	2,	11,	15,	12,	9,	7,	3,	10,	5,	0,
		15,	12,	8,	2,	4,	9,	1,	7,	5,	11,	3,	14,	10,	0,	6,	13,

		15,	1,	8,	14,	6,	11,	3,	4,	9,	7,	2,	13,	12,	0,	5,	10,
		3,	13,	4,	7,	15,	2,	8,	14,	12,	0,	1,	10,	6,	9,	11,	5,
		0,	14,	7,	11,	10,	4,	13,	1,	5,	8,	12,	6,	9,	3,	2,	15,
		13,	8,	10,	1,	3,	15,	4,	2,	11,	6,	7,	12,	0,	5,	14,	9,

		10,	0,	9,	14,	6,	3,	15,	5,	1,	13,	12,	7,	11,	4,	2,	8,
		13,	7,	0,	9,	3,	4,	6,	10,	2,	8,	5,	14,	12,	11,	15,	1,
		13,	6,	4,	9,	8,	15,	3,	0,	11,	1,	2,	12,	5,	10,	14,	7,
		1,	10,	13,	0,	6,	9,	8,	7,	4,	15,	14,	3,	11,	5,	2,	12,

		7,	13,	14,	3,	0,	6,	9,	10,	1,	2,	8,	5,	11,	12,	4,	15,
		13,	8,	11,	5,	6,	15,	0,	3,	4,	7,	2,	12,	1,	10,	14,	9,
		10,	6,	9,	0,	12,	11,	7,	13,	15,	1,	3,	14,	5,	2,	8,	4,
		3,	15,	0,	6,	10,	1,	13,	8,	9,	4,	5,	11,	12,	7,	2,	14,

		2,	12,	4,	1,	7,	10,	11,	6,	8,	5,	3,	15,	13,	0,	14,	9,
		14,	11,	2,	12,	4,	7,	13,	1,	5,	0,	15,	10,	3,	9,	8,	6,
		4,	2,	1,	11,	10,	13,	7,	8,	15,	9,	12,	5,	6,	3,	0,	14,
		11,	8,	12,	7,	1,	14,	2,	13,	6,	15,	0,	9,	10,	4,	5,	3,

		12,	1,	10,	15,	9,	2,	6,	8,	0,	13,	3,	4,	14,	7,	5,	11,
		10,	15,	4,	2,	7,	12,	9,	5,	6,	1,	13,	14,	0,	11,	3,	8,
		9,	14,	15,	5,	2,	8,	12,	3,	7,	0,	4,	10,	1,	13,	11,	6,
		4,	3,	2,	12,	9,	5,	15,	10,	11,	14,	1,	7,	6,	0,	8,	13,

		4,	11,	2,	14,	15,	0,	8,	13,	3,	12,	9,	7,	5,	10,	6,	1,
		13,	0,	11,	7,	4,	9,	1,	10,	14,	3,	5,	12,	2,	15,	8,	6,
		1,	4,	11,	13,	12,	3,	7,	14,	10,	15,	6,	8,	0,	5,	9,	2,
		6,	11,	13,	8,	1,	4,	10,	7,	9,	5,	0,	15,	14,	2,	3,	12,

		13,	2,	8,	4,	6,	15,	11,	1,	10,	9,	3,	14,	5,	0,	12,	7,
		1,	15,	13,	8,	10,	3,	7,	4,	12,	5,	6,	11,	0,	14,	9,	2,
		7,	11,	4,	1,	9,	12,	14,	2,	0,	6,	10,	13,	15,	3,	5,	8,
		2,	1,	14,	7,	4,	10,	8,	13,	15,	12,	9,	0,	3,	5,	6,	11,
	};

	DEVICE void Permute(const unsigned char* set, unsigned char* out, const unsigned char* table_div, const unsigned char* table_modshift, size_t table_size)
	{
		for (size_t i = 0; i < table_size / 8; i++)
		{
			out[i] = (!!(set[table_div[0]] & table_modshift[0]) << 0) |
				(!!(set[table_div[1]] & table_modshift[1]) << 1) |
				(!!(set[table_div[2]] & table_modshift[2]) << 2) |
				(!!(set[table_div[3]] & table_modshift[3]) << 3) |
				(!!(set[table_div[4]] & table_modshift[4]) << 4) |
				(!!(set[table_div[5]] & table_modshift[5]) << 5) |
				(!!(set[table_div[6]] & table_modshift[6]) << 6) |
				(!!(set[table_div[7]] & table_modshift[7]) << 7);

			table_div += 8;
			table_modshift += 8;
		}
	}

	void RotateLeft1Bit(unsigned char* set, size_t bit_start, size_t bit_size)
	{
		// First byte that we are modifying
		unsigned char byte_start = unsigned char(bit_start / 8);

		// First bit of first byte we are modifying
		unsigned char start_bit_index = bit_start % 8;

		unsigned char end_bit_index = ((bit_start + bit_size) % 8);

		// Saving the first bit that will be cut off
		bool first = (set[byte_start] >> start_bit_index) & 1;

		size_t i = byte_start;

		{
			// Mask to make sure we don't effect the bits outside the range
			unsigned char mask = 0 - (1 << start_bit_index);

			// Shift the bits in our range
			set[i] = ((set[i] >> 1) & mask) | (set[i] & ~mask);

			// Copy the first bit from the next byte to the last bit in this byte
			set[i] |= (set[i + 1] & 1) << 7;
		}

		for (i++; i < byte_start + (bit_size / 8); i++)
		{
			// Shift all bits in this byte
			set[i] = set[i] >> 1;

			// Copy the first bit from the next byte to the last bit in this byte
			set[i] |= (set[i + 1] & 1) << 7;
		}

		{
			// Mask to make sure we don't effect the bits outside the range
			unsigned char mask = end_bit_index ? (0 - (1 << end_bit_index)) : 0;

			// Shift the bits in our range
			set[i] = ((set[i] & ~mask) >> 1) | (set[i] & mask);

			// Copy the first bit from the first byte to the last bit in this byte
			set[i] |= first << ((end_bit_index + 7) % 8);
		}
	}

	void RotateLeft2Bit(unsigned char* set, size_t bit_start, size_t bit_size)
	{
		// First byte that we are modifying
		size_t byte_start = bit_start / 8;

		// First bit of first byte we are modifying
		unsigned char start_bit_index = bit_start % 8;

		unsigned char end_bit_index = ((bit_start + bit_size) % 8);

		// Saving the first 2 bits thats will be cut off
		unsigned char first = (set[byte_start] >> (bit_start % 8)) & 3;

		size_t i = byte_start;

		{
			// Mask to make sure we don't effect the bits outside the range
			unsigned char mask = 0 - (1 << start_bit_index);

			// Shift the bits in our range
			set[i] = ((set[i] >> 2) & mask) | (set[i] & ~mask);

			// Copy the first bit from the next byte to the last bit in this byte
			set[i] |= (set[i + 1] & 3) << 6;
		}

		for (i++; i < byte_start + (bit_size / 8); i++)
		{
			set[i] = set[i] >> 2;

			set[i] |= (set[i + 1] & 3) << 6;
		}

		{
			// Mask to make sure we don't effect the bits outside the range
			unsigned char mask = end_bit_index ? (0 - (1 << end_bit_index)) : 0;

			// Shift the bits in our range
			set[i] = ((set[i] & ~mask) >> 2) | (set[i] & mask);

			// Copy the first bit from the first byte to the last bit in this byte
			set[i] |= first << ((end_bit_index + 6) % 8);
		}
	}

	__forceinline__ __device__ unsigned char V(unsigned char byte, unsigned char from, unsigned char to)
	{
		from = 7 - from;

		if (to == from)
		{
			return byte & (1 << from);
		}
		else if (to > from)
		{
			return (byte & (1 << from)) << (to - from);
		}
		else
		{
			return (byte & (1 << from)) >> (from - to);
		}
	}

	__device__ void Compress(const unsigned char input[6], unsigned char output[4])
	{
		for (size_t i = 0; i < 2; i++)
		{
			const unsigned char* in = input + (i * 3);

			unsigned char* out = output + (i * 2);

			{
				unsigned char row = V(in[0], 0, 1) | V(in[0], 5, 0);
				unsigned char column = V(in[0], 1, 3) | V(in[0], 2, 2) | V(in[0], 3, 1) | V(in[0], 4, 0);

				unsigned char val = sbox[i][0][row][column];

				out[0] |= val << 4;
			}

			{
				unsigned char row = V(in[0], 6, 1) | V(in[1], 3, 0);
				unsigned char column = V(in[0], 7, 3) | V(in[1], 0, 2) | V(in[1], 1, 1) | V(in[1], 2, 0);

				unsigned char val = sbox[i][1][row][column];

				out[0] |= val;
			}

			{
				unsigned char row = V(in[1], 4, 1) | V(in[2], 1, 0);
				unsigned char column = V(in[1], 5, 3) | V(in[1], 6, 2) | V(in[1], 7, 1) | V(in[2], 0, 0);

				unsigned char val = sbox[i][2][row][column];

				out[1] |= val << 4;
			}

			{
				unsigned char row = V(in[2], 2, 1) | V(in[2], 7, 0);
				unsigned char column = V(in[2], 3, 3) | V(in[2], 4, 2) | V(in[2], 5, 1) | V(in[2], 6, 0);

				unsigned char val = sbox[i][3][row][column];

				out[1] |= val;
			}
		}
	}

	__device__ void CryptBlock(const unsigned char* subkeys, unsigned char* block)
	{
		unsigned char temp[8] = { 0 };

		Permute(block, temp, initial_perm_l, initial_perm_r, 64);

		unsigned char* left = temp;
		unsigned char* right = temp + 4;

		for (size_t key = 0; key < 16; key++)
		{
			const unsigned char* round_key = subkeys + (key * 6);

			unsigned char temp2[6] = { 0 };

			// Mangle
			{
				Permute(right, temp2, expansion_l, expansion_r, 48);

				for (size_t i = 0; i < 6; i++)
				{
					temp2[i] ^= round_key[i];
				}

				unsigned char temp3[4] = { 0 };

				Compress(temp2, temp3);

				Permute(temp3, temp2, pbox_l, pbox_r, 32);
			}

			for (size_t i = 0; i < 4; i++)
			{
				temp2[i] ^= left[i];
			}

			if (key < 15)
			{
				for (size_t i = 0; i < 4; i++)
				{
					left[i] = right[i];
				}

				for (size_t i = 0; i < 4; i++)
				{
					right[i] = temp2[i];
				}
			}
			else
			{
				for (size_t i = 0; i < 4; i++)
				{
					left[i] = temp2[i];
				}
			}
		}

		Permute(temp, block, final_perm_l, final_perm_r, 64);
	}

	__global__ void CryptBlocks(const unsigned char* subkeys, unsigned char* block)
	{
		block += (threadIdx.x * 8);

		CryptBlock(subkeys + (96 * 0), block);
	}

	__global__ void TripleCryptBlocksOld(const unsigned char* subkeys, unsigned char* block)
	{
		block += (threadIdx.x * 8);

		CryptBlock(subkeys + (96 * 0), block);
		CryptBlock(subkeys + (96 * 1), block);
		CryptBlock(subkeys + (96 * 2), block);
	}

	__global__ void TripleCryptBlocks(const unsigned char* subkeys, unsigned char* block, unsigned int group_size, unsigned int num_blocks)
	{
		unsigned int group = threadIdx.x;
		size_t group_start = group * group_size;
		size_t group_end = __min(group_start + group_size, num_blocks);

		for (size_t block_id = group_start; block_id < group_end; block_id++)
		{
			CryptBlock(subkeys + (96 * 0), block + (block_id * 8));
			CryptBlock(subkeys + (96 * 1), block + (block_id * 8));
			CryptBlock(subkeys + (96 * 2), block + (block_id * 8));
		}
	}

	TripleDES::TripleDES(const std::string& key, size_t group_size) :
		EncryptBase(key),
		m_group_size(group_size)
	{
		if (group_size == 0 || group_size > (SIZE_MAX / k_block_size))
		{
			throw Exception{};
		}

		if (key.size() != k_min_key_size)
		{
			throw Exception{};
		}

		if (hipSetDevice(0) != hipSuccess)
		{
			fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
			throw Exception{};
		}

		if (hipMalloc(&m_subkeys, 6 * 96) != hipSuccess)
		{
			fprintf(stderr, "hipMalloc failed!");
			throw Exception{};
		}

		unsigned char subkeys[6][96] = { 0 };

		const unsigned char enc_pos[3] = { 0, 4, 2 };
		const unsigned char dec_pos[3] = { 3, 1, 5 };

		for (size_t k = 0; k < 3; k++)
		{
			unsigned char* enc_subkeys = subkeys[enc_pos[k]];
			unsigned char* dec_subkeys = subkeys[dec_pos[k]] + (6 * 15);

			unsigned char temp[7] = { 0 };
			Permute((const unsigned char*)key.data() + (k * 8), temp, key_perm_l, key_perm_r, 56);

			for (size_t i = 0; i < 16; i++)
			{
				switch (i)
				{
				case 0:
				case 1:
				case 8:
				case 15:
					RotateLeft1Bit(temp, 0, 28);
					RotateLeft1Bit(temp, 28, 28);
					break;

				default:
					RotateLeft2Bit(temp, 0, 28);
					RotateLeft2Bit(temp, 28, 28);
					break;
				}

				Permute(temp, enc_subkeys, left_round_perm_l, left_round_perm_r, 24);

				Permute(temp, enc_subkeys + 3, right_round_perm_l, right_round_perm_r, 24);

				Permute(temp, dec_subkeys, left_round_perm_l, left_round_perm_r, 24);

				Permute(temp, dec_subkeys + 3, right_round_perm_l, right_round_perm_r, 24);

				enc_subkeys += 6;
				dec_subkeys -= 6;
			}
		}

		if (hipMemcpy(m_subkeys, subkeys, 6 * 96, hipMemcpyHostToDevice) != hipSuccess)
		{
			fprintf(stderr, "hipMemcpy failed!");
			throw Exception{};
		}
	}

	TripleDES::~TripleDES()
	{
		hipFree(m_subkeys);
	}

	std::string TripleDES::GetName() const
	{
		char buffer[32];
		sprintf_s(buffer, "Triple DES CPU %zi per group", m_group_size);
		return buffer;
	}

	void TripleDES::EncryptInPlace(std::string& input) const
	{
		if (input.size() == 0 || input.size() % k_block_size != 0)
		{
			throw Exception{};
		}

		unsigned char* mem = 0;

		if (hipMalloc(&mem, input.size()) != hipSuccess)
		{
			fprintf(stderr, "hipMalloc failed!");
			throw Exception{};
		}

		if (hipMemcpy(mem, input.data(), input.size(), hipMemcpyHostToDevice) != hipSuccess)
		{
			fprintf(stderr, "hipMemcpy failed!");
			throw Exception{};
		}

		unsigned int num_blocks = (unsigned int)input.size() / k_block_size;
		unsigned int num_iterations = (num_blocks + m_group_size - 1) / m_group_size;
		TripleCryptBlocks<<<1, num_iterations>>>(m_subkeys, mem, m_group_size, num_blocks);

		if (hipMemcpy(input.data(), mem, input.size(), hipMemcpyDeviceToHost) != hipSuccess)
		{
			fprintf(stderr, "hipMemcpy failed!");
			throw Exception{};
		}

		if (hipFree(mem) != hipSuccess)
		{
			fprintf(stderr, "hipFree failed!");
			throw Exception{};
		}
	}

	void TripleDES::DecryptInPlace(std::string& input) const
	{
		if (input.size() == 0 || input.size() % k_block_size != 0)
		{
			throw Exception{};
		}

		unsigned char* mem = 0;

		if (hipMalloc(&mem, input.size()) != hipSuccess)
		{
			fprintf(stderr, "hipMalloc failed!");
			throw Exception{};
		}

		if (hipMemcpy(mem, input.data(), input.size(), hipMemcpyHostToDevice) != hipSuccess)
		{
			fprintf(stderr, "hipMemcpy failed!");
			throw Exception{};
		}

		unsigned int num_blocks = (unsigned int)input.size() / k_block_size;
		unsigned int num_iterations = (num_blocks + m_group_size - 1) / m_group_size;
		TripleCryptBlocks<<<1, num_iterations>>>(m_subkeys + (96 * 3), mem, m_group_size, num_blocks);

		if (hipMemcpy(input.data(), mem, input.size(), hipMemcpyDeviceToHost) != hipSuccess)
		{
			fprintf(stderr, "hipMemcpy failed!");
			throw Exception{};
		}

		if (hipFree(mem) != hipSuccess)
		{
			fprintf(stderr, "hipFree failed!");
			throw Exception{};
		}
	}
}